#include "hip/hip_runtime.h"
// Copyright 2014-2017 Oxford University Innovation Limited and the authors of InfiniTAM

#include "ITMLowLevelEngine_CUDA.h"

#include "../Shared/ITMLowLevelEngine_Shared.h"
#include <ITMLib/Utils/ITMProjectionUtils.h>
#include <Utils/ITMCUDAUtils.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/device_ptr.h>

#include <ORUtils/CUDADefines.h>

using namespace ITMLib;

ITMLowLevelEngine_CUDA::ITMLowLevelEngine_CUDA()
{
	ORcudaSafeCall(hipMalloc((void**) &counterTempData_device, sizeof(int)));
	ORcudaSafeCall(hipHostMalloc((void**) &counterTempData_host, sizeof(int)));
}

ITMLowLevelEngine_CUDA::~ITMLowLevelEngine_CUDA()
{
	ORcudaSafeCall(hipFree(counterTempData_device));
	ORcudaSafeCall(hipHostFree(counterTempData_host));
}

__global__ void convertColourToIntensity_device(float* imageData_out, Vector2i dims, const Vector4u* imageData_in);

__global__ void boxFilter2x2_device(float* imageData_out, const float* imageData_in, Vector2i dims);

__global__ void
filterSubsample_device(float* imageData_out, Vector2i newDims, const float* imageData_in, Vector2i oldDims);

__global__ void
filterSubsample_device(Vector4u* imageData_out, Vector2i newDims, const Vector4u* imageData_in, Vector2i oldDims);

__global__ void
filterSubsampleWithHoles_device(float* imageData_out, Vector2i newDims, const float* imageData_in, Vector2i oldDims);

__global__ void filterSubsampleWithHoles_device(Vector4f* imageData_out, Vector2i newDims, const Vector4f* imageData_in,
                                                Vector2i oldDims);

__global__ void gradientX_device(Vector4s* grad, const Vector4u* image, Vector2i imgSize);

__global__ void gradientY_device(Vector4s* grad, const Vector4u* image, Vector2i imgSize);

__global__ void gradientXY_device(Vector2f* grad, const float* image, Vector2i imgSize);

__global__ void countValidDepths_device(const float* imageData_in, int imgSizeTotal, int* counterTempData_device);

struct PointCloudAccumulator
{
	int noPoints;
	Vector3f pointSum;
};

__global__
void
computePointCloudCenter_device(PointCloudAccumulator* accumulator, const Vector4f* cloud, Vector2i imageSize);

__global__
void computeDepthCloudCenter_device(PointCloudAccumulator* accumulator, const float* depth, Vector2i imageSize,
                                    Vector4f intrinsics);

// host methods

void ITMLowLevelEngine_CUDA::CopyImage(ITMUChar4Image* image_out, const ITMUChar4Image* image_in) const
{
	Vector4u* dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u* src = image_in->GetData(MEMORYDEVICE_CUDA);

	ORcudaSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(Vector4u), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::CopyImage(ITMFloatImage* image_out, const ITMFloatImage* image_in) const
{
	float* dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const float* src = image_in->GetData(MEMORYDEVICE_CUDA);

	ORcudaSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(float), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::CopyImage(ITMFloat4Image* image_out, const ITMFloat4Image* image_in) const
{
	Vector4f* dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4f* src = image_in->GetData(MEMORYDEVICE_CUDA);

	ORcudaSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(Vector4f), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::ConvertColourToIntensity(ITMFloatImage* image_out, const ITMUChar4Image* image_in) const
{
	const Vector2i dims = image_in->noDims;
	image_out->ChangeDims(dims);

	float* dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u* src = image_in->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) dims.x / (float) blockSize.x), (int) ceil((float) dims.y / (float) blockSize.y));

	convertColourToIntensity_device << < gridSize, blockSize >> >(dest, dims, src);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::FilterIntensity(ITMFloatImage* image_out, const ITMFloatImage* image_in) const
{
	Vector2i dims = image_in->noDims;

	image_out->ChangeDims(dims);
	image_out->Clear(0);

	const float* imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float* imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) dims.x / (float) blockSize.x), (int) ceil((float) dims.y / (float) blockSize.y));

	boxFilter2x2_device << < gridSize, blockSize >> >(imageData_out, imageData_in, dims);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::FilterSubsample(ITMUChar4Image* image_out, const ITMUChar4Image* image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims;
	newDims.x = image_in->noDims.x / 2;
	newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const Vector4u* imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	Vector4u* imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) newDims.x / (float) blockSize.x),
	              (int) ceil((float) newDims.y / (float) blockSize.y));

	filterSubsample_device << < gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::FilterSubsample(ITMFloatImage* image_out, const ITMFloatImage* image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims;
	newDims.x = image_in->noDims.x / 2;
	newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);
	image_out->Clear(0);

	const float* imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float* imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) newDims.x / (float) blockSize.x),
	              (int) ceil((float) newDims.y / (float) blockSize.y));

	filterSubsample_device << < gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::FilterSubsampleWithHoles(ITMFloatImage* image_out, const ITMFloatImage* image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims;
	newDims.x = image_in->noDims.x / 2;
	newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const float* imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float* imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) newDims.x / (float) blockSize.x),
	              (int) ceil((float) newDims.y / (float) blockSize.y));

	filterSubsampleWithHoles_device << < gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::FilterSubsampleWithHoles(ITMFloat4Image* image_out, const ITMFloat4Image* image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims;
	newDims.x = image_in->noDims.x / 2;
	newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const Vector4f* imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	Vector4f* imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) newDims.x / (float) blockSize.x),
	              (int) ceil((float) newDims.y / (float) blockSize.y));

	filterSubsampleWithHoles_device << < gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::GradientX(ITMShort4Image* grad_out, const ITMUChar4Image* image_in) const
{
	grad_out->ChangeDims(image_in->noDims);
	Vector2i imgSize = image_in->noDims;

	Vector4s* grad = grad_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u* image = image_in->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) imgSize.x / (float) blockSize.x),
	              (int) ceil((float) imgSize.y / (float) blockSize.y));

	ORcudaSafeCall(hipMemset(grad, 0, imgSize.x * imgSize.y * sizeof(Vector4s)));

	gradientX_device << < gridSize, blockSize >> >(grad, image, imgSize);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::GradientY(ITMShort4Image* grad_out, const ITMUChar4Image* image_in) const
{
	grad_out->ChangeDims(image_in->noDims);
	Vector2i imgSize = image_in->noDims;

	Vector4s* grad = grad_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u* image = image_in->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) imgSize.x / (float) blockSize.x),
	              (int) ceil((float) imgSize.y / (float) blockSize.y));

	ORcudaSafeCall(hipMemset(grad, 0, imgSize.x * imgSize.y * sizeof(Vector4s)));

	gradientY_device << < gridSize, blockSize >> >(grad, image, imgSize);
	ORcudaKernelCheck;
}

void ITMLowLevelEngine_CUDA::GradientXY(ITMFloat2Image* grad_out, const ITMFloatImage* image_in) const
{
	Vector2i imgSize = image_in->noDims;
	grad_out->ChangeDims(imgSize);
	grad_out->Clear();

	Vector2f* grad = grad_out->GetData(MEMORYDEVICE_CUDA);
	const float* image = image_in->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) imgSize.x / (float) blockSize.x),
	              (int) ceil((float) imgSize.y / (float) blockSize.y));

	gradientXY_device << < gridSize, blockSize >> >(grad, image, imgSize);
	ORcudaKernelCheck;
}

int ITMLowLevelEngine_CUDA::CountValidDepths(const ITMFloatImage* image_in) const
{
	const float* imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	Vector2i imgSize = image_in->noDims;

	dim3 blockSize(256);
	dim3 gridSize((int) ceil((float) imgSize.x * imgSize.y / (float) blockSize.x));

	ORcudaSafeCall(hipMemset(counterTempData_device, 0, sizeof(int)));
	countValidDepths_device <<<gridSize, blockSize>>>(imageData_in, imgSize.x * imgSize.y, counterTempData_device);
	ORcudaKernelCheck;
	ORcudaSafeCall(hipMemcpy(counterTempData_host, counterTempData_device, sizeof(int), hipMemcpyDeviceToHost));

	return *counterTempData_host;
}

void ITMLowLevelEngine_CUDA::ComputePointCloudCenter(Vector3f& center, size_t& noValidPoints,
                                                     const ITMFloat4Image* cloud) const
{
	PointCloudAccumulator* accumulator_device;
	ORcudaSafeCall(hipMalloc((void**) &accumulator_device, sizeof(PointCloudAccumulator)));
	ORcudaSafeCall(hipMemset(accumulator_device, 0, sizeof(PointCloudAccumulator)));

	Vector2i imageSize = cloud->noDims;

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) imageSize.x / (float) blockSize.x),
	              (int) ceil((float) imageSize.y / (float) blockSize.y));
	computePointCloudCenter_device <<<gridSize, blockSize>>>(accumulator_device, cloud->GetData(MEMORYDEVICE_CUDA),
	                                                         imageSize);

	PointCloudAccumulator accumulator_host;
	ORcudaSafeCall(
		hipMemcpy(&accumulator_host, accumulator_device, sizeof(PointCloudAccumulator), hipMemcpyDeviceToHost));
	noValidPoints = accumulator_host.noPoints;
	center = accumulator_host.pointSum;
	if (noValidPoints > 0) center /= noValidPoints;

	ORcudaSafeCall(hipFree(accumulator_device));
}

void
ITMLowLevelEngine_CUDA::ComputeDepthCloudCenter(Vector3f& center, size_t& noValidPoints, const ITMFloatImage* depth,
                                                Vector4f intrinsics) const
{
	PointCloudAccumulator* accumulator_device;
	ORcudaSafeCall(hipMalloc((void**) &accumulator_device, sizeof(PointCloudAccumulator)));
	ORcudaSafeCall(hipMemset(accumulator_device, 0, sizeof(PointCloudAccumulator)));

	Vector2i imageSize = depth->noDims;

	dim3 blockSize(16, 16);
	dim3 gridSize((int) ceil((float) imageSize.x / (float) blockSize.x),
	              (int) ceil((float) imageSize.y / (float) blockSize.y));

	computeDepthCloudCenter_device <<<gridSize, blockSize>>>(accumulator_device, depth->GetData(MEMORYDEVICE_CUDA),
	                                                         imageSize, invertProjectionParams(intrinsics));

	PointCloudAccumulator accumulator_host;
	ORcudaSafeCall(
		hipMemcpy(&accumulator_host, accumulator_device, sizeof(PointCloudAccumulator), hipMemcpyDeviceToHost));
	noValidPoints = accumulator_host.noPoints;
	center = accumulator_host.pointSum;
	if (noValidPoints > 0) center /= noValidPoints;

	ORcudaSafeCall(hipFree(accumulator_device));
}

void ITMLowLevelEngine_CUDA::RescaleDepthImage(ITMFloatImage* image, float factor) const
{
	thrust::device_ptr<float> ptr = thrust::device_pointer_cast(image->GetData(MEMORYDEVICE_CUDA));
	thrust::transform(ptr, ptr + image->dataSize, ptr, factor * thrust::placeholders::_1);
}

// device functions

__global__
void computePointCloudCenter_device(PointCloudAccumulator* accumulator, const Vector4f* cloud, const Vector2i imageSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = PixelCoordsToIndex(x, y, imageSize);

	int locId_local = threadIdx.x + threadIdx.y * blockDim.x;

	__shared__ bool blockHasValidPoint;

	blockHasValidPoint = false;
	__syncthreads();

	Vector3f point(0, 0, 0);
	bool isValidPoint = false;

	if (x < imageSize.width && y < imageSize.height)
	{
		const Vector4f& point_ = cloud[idx];
		if (point_.w >= 0.f && point_.z >= 1e-3f)
		{
			point = point_.toVector3();
			blockHasValidPoint = true;
			isValidPoint = true;
		}
	}

	__syncthreads();
	if (!blockHasValidPoint) return;

	parallelReduceAtomic<256>(accumulator->noPoints, (int) isValidPoint, locId_local);
	parallelReduceVector3<256>(accumulator->pointSum, point, locId_local);
}

__global__
void computeDepthCloudCenter_device(PointCloudAccumulator* accumulator, const float* depth,
                                    const Vector2i imageSize, const Vector4f invProjParams)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = PixelCoordsToIndex(x, y, imageSize);

	int locId_local = threadIdx.x + threadIdx.y * blockDim.x;

	__shared__ bool blockHasValidPoint;

	blockHasValidPoint = false;
	__syncthreads();

	Vector3f point(0, 0, 0);
	bool isValidPoint = false;

	if (x < imageSize.width && y < imageSize.height)
	{
		const float depthValue = depth[idx];
		if (depthValue > 1e-3)
		{
			point = reprojectImagePoint(x, y, depthValue, invProjParams);
			blockHasValidPoint = true;
			isValidPoint = true;
		}
	}

	__syncthreads();
	if (!blockHasValidPoint) return;

	parallelReduceAtomic<256>(accumulator->noPoints, (int) isValidPoint, locId_local);
	parallelReduceVector3<256>(accumulator->pointSum, point, locId_local);
}

__global__ void convertColourToIntensity_device(float* imageData_out, Vector2i dims, const Vector4u* imageData_in)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > dims.x - 1 || y > dims.y - 1) return;

	convertColourToIntensity(imageData_out, x, y, dims, imageData_in);
}

__global__ void boxFilter2x2_device(float* imageData_out, const float* imageData_in, Vector2i dims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= dims.x - 2 || y >= dims.y - 2 || x <= 1 || y <= 1) return;

	boxFilter2x2(imageData_out, x, y, dims, imageData_in, x, y, dims);
}

__global__ void
filterSubsample_device(Vector4u* imageData_out, Vector2i newDims, const Vector4u* imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsample(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void
filterSubsample_device(float* imageData_out, Vector2i newDims, const float* imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 2 || y > newDims.y - 2 || x < 1 || y < 1) return;

	boxFilter2x2(imageData_out, x, y, newDims, imageData_in, x * 2, y * 2, oldDims);
}

__global__ void
filterSubsampleWithHoles_device(float* imageData_out, Vector2i newDims, const float* imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsampleWithHoles(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void filterSubsampleWithHoles_device(Vector4f* imageData_out, Vector2i newDims, const Vector4f* imageData_in,
                                                Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsampleWithHoles(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void gradientX_device(Vector4s* grad, const Vector4u* image, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 1 || x > imgSize.x - 2 || y < 1 || y > imgSize.y - 2) return;

	gradientX(grad, x, y, image, imgSize);
}

__global__ void gradientY_device(Vector4s* grad, const Vector4u* image, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 1 || x > imgSize.x - 2 || y < 1 || y > imgSize.y - 2) return;

	gradientY(grad, x, y, image, imgSize);
}

__global__ void gradientXY_device(Vector2f* grad, const float* image, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 1 || x > imgSize.x - 2 || y < 1 || y > imgSize.y - 2) return;

	gradientXY(grad, x, y, image, imgSize);
}

__global__ void countValidDepths_device(const float* imageData_in, int imgSizeTotal, int* counterTempData_device)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int locId_local = threadIdx.x;

	__shared__ int dim_shared[256];
	//__shared__ bool should_prefix;

	//should_prefix = false;
	//__syncthreads();

	bool isValidPoint = false;

	if (i < imgSizeTotal)
	{
		if (imageData_in[i] > 0.0f) isValidPoint = true;
	}

	//__syncthreads();
	//if (!should_prefix) return;

	dim_shared[locId_local] = isValidPoint;
	__syncthreads();

	if (locId_local < 128) dim_shared[locId_local] += dim_shared[locId_local + 128];
	__syncthreads();
	if (locId_local < 64) dim_shared[locId_local] += dim_shared[locId_local + 64];
	__syncthreads();

	if (locId_local < 32) warpReduce(dim_shared, locId_local);

	if (locId_local == 0) atomicAdd(counterTempData_device, dim_shared[locId_local]);
}

