#include "hip/hip_runtime.h"
// Copyright 2014-2017 Oxford University Innovation Limited and the authors of InfiniTAM

#include "ITMViewBuilder_CUDA.h"

#include "ITMLib/Engines/ViewBuilding/Shared/ITMViewBuilder_Shared.h"
#include "ORUtils/CUDADefines.h"
#include "ORUtils/MemoryBlock.h"
#include "ITMLib/Utils/ITMTimer.h"

using namespace ITMLib;
using namespace ORUtils;

ITMViewBuilder_CUDA::ITMViewBuilder_CUDA(const ITMRGBDCalib& calib):ITMViewBuilder(calib) { }
ITMViewBuilder_CUDA::~ITMViewBuilder_CUDA(void) { }

//---------------------------------------------------------------------------
//
// kernel function declaration 
//
//---------------------------------------------------------------------------


__global__ void convertDisparityToDepth_device(float *depth_out, const short *depth_in, Vector2f disparityCalibParams, float fx_depth, Vector2i imgSize);
__global__ void convertDepthAffineToFloat_device(float *d_out, const short *d_in, Vector2i imgSize, Vector2f depthCalibParams);
__global__ void filterDepth_device(float *imageData_out, const float *imageData_in, Vector2i imgDims);
__global__ void filterNormals_device(Vector4f *normals_out, const Vector4f *normals_in, Vector2i imgDims);
__global__ void ComputeNormalAndWeight_device(const float* depth_in, Vector4f* normal_out, float *sigmaL_out, Vector2i imgDims, Vector4f intrinsic);

//---------------------------------------------------------------------------
//
// host methods
//
//---------------------------------------------------------------------------

void ITMViewBuilder_CUDA::UpdateView(ITMView **view_ptr, ITMUChar4Image *rgbImage, ITMShortImage *rawDepthImage, bool useBilateralFilter, bool modelSensorNoise, bool storePreviousImage)
{
	timeStats.Reset();
	ITMTimer timer;
	timer.Tick();

	if (*view_ptr == nullptr)
	{
		*view_ptr = new ITMView(calib, rgbImage->noDims, rawDepthImage->noDims, true);
		delete this->shortImage;
		this->shortImage = new ITMShortImage(rawDepthImage->noDims, true, true);
		delete this->floatImage;
		this->floatImage = new ITMFloatImage(rawDepthImage->noDims, true, true);
		delete this->normals;
		this->normals = new ITMFloat4Image(rawDepthImage->noDims, true, true);

		if (modelSensorNoise)
		{
			(*view_ptr)->depthNormal = new ITMFloat4Image(rawDepthImage->noDims, true, true);
			(*view_ptr)->depthUncertainty = new ITMFloatImage(rawDepthImage->noDims, true, true);
		}
	}

	ITMView *view = *view_ptr;

	if (storePreviousImage)
	{
		if (!view->rgb_prev) view->rgb_prev = new ITMUChar4Image(rgbImage->noDims, true, true);
		else view->rgb_prev->SetFrom(view->rgb, MemoryBlock<Vector4u>::CUDA_TO_CUDA);
	}

	view->rgb->SetFrom(rgbImage, MemoryBlock<Vector4u>::CPU_TO_CUDA);
	this->shortImage->SetFrom(rawDepthImage, MemoryBlock<short>::CPU_TO_CUDA);

	switch (view->calib.disparityCalib.GetType())
	{
	case ITMDisparityCalib::TRAFO_KINECT:
		this->ConvertDisparityToDepth(view->depth, this->shortImage, &(view->calib.intrinsics_d), view->calib.disparityCalib.GetParams());
		break;
	case ITMDisparityCalib::TRAFO_AFFINE:
		this->ConvertDepthAffineToFloat(view->depth, this->shortImage, view->calib.disparityCalib.GetParams());
		break;
	default:
		break;
	}
	timeStats.copyImages += timer.Tock();

	if (useBilateralFilter)
	{
		timer.Tick();
		//5 steps of bilateral filtering
		this->DepthFiltering(this->floatImage, view->depth);
		this->DepthFiltering(view->depth, this->floatImage);
		this->DepthFiltering(this->floatImage, view->depth);
		this->DepthFiltering(view->depth, this->floatImage);
		this->DepthFiltering(this->floatImage, view->depth);
		view->depth->SetFrom(this->floatImage, MemoryBlock<float>::CUDA_TO_CUDA);
		timeStats.bilateralFilter = timer.Tock();
	}

	if (modelSensorNoise)
	{
		timer.Tick();
//#define FILTER_NORMALS
#ifdef FILTER_NORMALS
		this->ComputeNormalAndWeights(this->normals, view->depthUncertainty, view->depth,
		                              view->calib.intrinsics_d.projectionParamsSimple.all);
		this->NormalFiltering(view->depthNormal, this->normals);
#else
		// normals from filteres image
		this->ComputeNormalAndWeights(view->depthNormal, view->depthUncertainty, view->depth,
																	view->calib.intrinsics_d.projectionParamsSimple.all);
#endif
		timeStats.normalEstimation = timer.Tock();
	}
}

void ITMViewBuilder_CUDA::UpdateView(ITMView **view_ptr, ITMUChar4Image *rgbImage, ITMShortImage *depthImage, bool useBilateralFilter, ITMIMUMeasurement *imuMeasurement, bool modelSensorNoise, bool storePreviousImage)
{
	if (*view_ptr == nullptr)
	{
		*view_ptr = new ITMViewIMU(calib, rgbImage->noDims, depthImage->noDims, true);
		delete this->shortImage;
		this->shortImage = new ITMShortImage(depthImage->noDims, true, true);
		delete this->floatImage;
		this->floatImage = new ITMFloatImage(depthImage->noDims, true, true);
		delete this->normals;
		this->normals = new ITMFloat4Image(depthImage->noDims, true, true);

		if (modelSensorNoise)
		{
			(*view_ptr)->depthNormal = new ITMFloat4Image(depthImage->noDims, true, true);
			(*view_ptr)->depthUncertainty = new ITMFloatImage(depthImage->noDims, true, true);
		}
	}

	ITMViewIMU* imuView = (ITMViewIMU*)(*view_ptr);
	imuView->imu->SetFrom(imuMeasurement);

	this->UpdateView(view_ptr, rgbImage, depthImage, useBilateralFilter, modelSensorNoise, storePreviousImage);
}

void ITMViewBuilder_CUDA::ConvertDisparityToDepth(ITMFloatImage *depth_out, const ITMShortImage *depth_in, const ITMIntrinsics *depthIntrinsics,
	Vector2f disparityCalibParams)
{
	Vector2i imgSize = depth_in->noDims;

	const short *d_in = depth_in->GetData(MEMORYDEVICE_CUDA);
	float *d_out = depth_out->GetData(MEMORYDEVICE_CUDA);

	float fx_depth = depthIntrinsics->projectionParamsSimple.fx;

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	convertDisparityToDepth_device << <gridSize, blockSize >> >(d_out, d_in, disparityCalibParams, fx_depth, imgSize);
	ORcudaKernelCheck;
}

void ITMViewBuilder_CUDA::ConvertDepthAffineToFloat(ITMFloatImage *depth_out, const ITMShortImage *depth_in, Vector2f depthCalibParams)
{
	Vector2i imgSize = depth_in->noDims;

	const short *d_in = depth_in->GetData(MEMORYDEVICE_CUDA);
	float *d_out = depth_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	convertDepthAffineToFloat_device << <gridSize, blockSize >> >(d_out, d_in, imgSize, depthCalibParams);
	ORcudaKernelCheck;
}

void ITMViewBuilder_CUDA::DepthFiltering(ITMFloatImage *image_out, const ITMFloatImage *image_in)
{
	Vector2i imgDims = image_in->noDims;

	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgDims.x / (float)blockSize.x), (int)ceil((float)imgDims.y / (float)blockSize.y));

	filterDepth_device << <gridSize, blockSize >> >(imageData_out, imageData_in, imgDims);
	ORcudaKernelCheck;
}

void ITMViewBuilder_CUDA::ComputeNormalAndWeights(ITMFloat4Image *normal_out, ITMFloatImage *sigmaZ_out, const ITMFloatImage *depth_in, Vector4f intrinsic)
{
	Vector2i imgDims = depth_in->noDims;

	const float *depthData_in = depth_in->GetData(MEMORYDEVICE_CUDA);

	float *sigmaZData_out = sigmaZ_out->GetData(MEMORYDEVICE_CUDA);
	Vector4f *normalData_out = normal_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgDims.x / (float)blockSize.x), (int)ceil((float)imgDims.y / (float)blockSize.y));

	ComputeNormalAndWeight_device << <gridSize, blockSize >> >(depthData_in, normalData_out, sigmaZData_out, imgDims, intrinsic);
	ORcudaKernelCheck;
}

void ITMViewBuilder_CUDA::NormalFiltering(ITMFloat4Image* normals_out, const ITMFloat4Image* normals_in)
{
	Vector2i imgDims = normals_in->noDims;

	const Vector4f *n_in = normals_in->GetData(MEMORYDEVICE_CUDA);
	Vector4f *n_out = normals_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgDims.x / (float)blockSize.x), (int)ceil((float)imgDims.y / (float)blockSize.y));

	filterNormals_device << <gridSize, blockSize >> >(n_out, n_in, imgDims);
	ORcudaKernelCheck;
}

//---------------------------------------------------------------------------
//
// kernel function implementation
//
//---------------------------------------------------------------------------

__global__ void convertDisparityToDepth_device(float *d_out, const short *d_in, Vector2f disparityCalibParams, float fx_depth, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= imgSize.x) || (y >= imgSize.y)) return;

	convertDisparityToDepth(d_out, x, y, d_in, disparityCalibParams, fx_depth, imgSize);
}

__global__ void convertDepthAffineToFloat_device(float *d_out, const short *d_in, Vector2i imgSize, Vector2f depthCalibParams)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= imgSize.x) || (y >= imgSize.y)) return;

	convertDepthAffineToFloat(d_out, x, y, d_in, imgSize, depthCalibParams);
}

__global__ void filterDepth_device(float *imageData_out, const float *imageData_in, Vector2i imgDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 2 || x > imgDims.x - 2 || y < 2 || y > imgDims.y - 2) return;

	filterDepth(imageData_out, imageData_in, x, y, imgDims);
}

__global__ void filterNormals_device(Vector4f *normals_out, const Vector4f *normals_in, Vector2i imgDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= imgDims.x || y >= imgDims.y) return;

	filterNormals(normals_out, normals_in, 2.5, 5.0, x, y, imgDims);
}

__global__ void ComputeNormalAndWeight_device(const float* depth_in, Vector4f* normal_out, float *sigmaZ_out, Vector2i imgDims, Vector4f intrinsic)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= imgDims.x || y >= imgDims.y)
		return;

	computeNormalAndWeight(depth_in, normal_out, sigmaZ_out, x, y, imgDims, intrinsic);
}

