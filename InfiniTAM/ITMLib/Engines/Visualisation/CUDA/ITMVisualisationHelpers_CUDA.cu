#include "hip/hip_runtime.h"
// Copyright 2014-2017 Oxford University Innovation Limited and the authors of InfiniTAM

#include "ITMVisualisationHelpers_CUDA.h"
#include "Engines/Reconstruction/Interface/ITMSceneReconstructionEngine.h"

//device implementations

namespace ITMLib
{

__global__ void countVisibleBlocks_device(const int *visibleEntryIDs, int noVisibleEntries, const ITMHashEntry *hashTable, uint *noBlocks, int minBlockId, int maxBlockId)
{
	int globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (globalIdx >= noVisibleEntries) return;

	int entryId = visibleEntryIDs[globalIdx];
	int blockId = hashTable[entryId].ptr;
	if ((blockId >= minBlockId) && (blockId <= maxBlockId)) atomicAdd(noBlocks, 1);
}

__global__ void buildCompleteVisibleList_device(const ITMHashEntry *hashTable, /*ITMHashCacheState *cacheStates, bool useSwapping,*/ int noTotalEntries,
	int *visibleEntryIDs, int *noVisibleEntries, HashEntryVisibilityType *entriesVisibleType, Matrix4f M, Vector4f projParams, Vector2i imgSize, float voxelSize)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	__shared__ bool shouldPrefix;

	unsigned char hashVisibleType = 0; //entriesVisibleType[targetIdx];
	const ITMHashEntry &hashEntry = hashTable[targetIdx];

	shouldPrefix = false;
	__syncthreads();

	if (hashEntry.ptr >= 0)
	{
		shouldPrefix = true;

		bool isVisible, isVisibleEnlarged;
		checkBlockVisibility<false>(isVisible, isVisibleEnlarged, hashEntry.pos, M, projParams, voxelSize, imgSize);

		hashVisibleType = isVisible;
	}

	if (hashVisibleType > 0) shouldPrefix = true;

	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType > 0, noVisibleEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) visibleEntryIDs[offset] = targetIdx;
	}
}

__global__ void projectAndSplitBlocks_device(const ITMHashEntry *hashEntries, const int *visibleEntryIDs, int noVisibleEntries,
	const Matrix4f pose_M, const Vector4f intrinsics, const Vector2i imgSize, float voxelSize, RenderingBlock *renderingBlocks,
	uint *noTotalBlocks)
{
	int in_offset = threadIdx.x + blockDim.x * blockIdx.x;

	const ITMHashEntry & blockData(hashEntries[visibleEntryIDs[in_offset]]);

	Vector2i upperLeft, lowerRight;
	Vector2f zRange;
	bool validProjection = false;
	if (in_offset < noVisibleEntries) if (blockData.ptr >= 0)
		validProjection = ProjectSingleBlock(blockData.pos, pose_M, intrinsics, imgSize, voxelSize, upperLeft, lowerRight, zRange);

	Vector2i requiredRenderingBlocks(ceilf((float)(lowerRight.x - upperLeft.x + 1) / renderingBlockSizeX),
		ceilf((float)(lowerRight.y - upperLeft.y + 1) / renderingBlockSizeY));

	size_t requiredNumBlocks = requiredRenderingBlocks.x * requiredRenderingBlocks.y;
	if (!validProjection) requiredNumBlocks = 0;

	int out_offset = computePrefixSum_device<uint>(requiredNumBlocks, noTotalBlocks, blockDim.x, threadIdx.x);
	if (!validProjection) return;
	if ((out_offset == -1) || (out_offset + requiredNumBlocks > MAX_RENDERING_BLOCKS)) return;

	CreateRenderingBlocks(renderingBlocks, out_offset, upperLeft, lowerRight, zRange);
}

__global__ void checkProjectAndSplitBlocks_device(const ITMHashEntry *hashEntries, int noHashEntries,
	const Matrix4f pose_M, const Vector4f intrinsics, const Vector2i imgSize, float voxelSize, RenderingBlock *renderingBlocks,
	uint *noTotalBlocks)
{
	int targetIdx = threadIdx.x + blockDim.x * blockIdx.x;
	if (targetIdx >= noHashEntries) return;

	const ITMHashEntry & hashEntry = hashEntries[targetIdx];

	Vector2i upperLeft, lowerRight;
	Vector2f zRange;
	bool validProjection = false;
	if (hashEntry.ptr >= 0) validProjection = ProjectSingleBlock(hashEntry.pos, pose_M, intrinsics, imgSize, voxelSize, upperLeft, lowerRight, zRange);

	Vector2i requiredRenderingBlocks(ceilf((float)(lowerRight.x - upperLeft.x + 1) / renderingBlockSizeX),
		ceilf((float)(lowerRight.y - upperLeft.y + 1) / renderingBlockSizeY));
	size_t requiredNumBlocks = requiredRenderingBlocks.x * requiredRenderingBlocks.y;
	if (!validProjection) requiredNumBlocks = 0;

	int out_offset = computePrefixSum_device<uint>(requiredNumBlocks, noTotalBlocks, blockDim.x, threadIdx.x);
	if (requiredNumBlocks == 0) return;
	if ((out_offset == -1) || (out_offset + requiredNumBlocks > MAX_RENDERING_BLOCKS)) return;

	CreateRenderingBlocks(renderingBlocks, out_offset, upperLeft, lowerRight, zRange);
}

__global__ void fillBlocks_device(uint noTotalBlocks, const RenderingBlock *renderingBlocks,
	Vector2i imgSize, Vector2f *minmaxData)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	int block = blockIdx.x * 4 + blockIdx.y;
	if (block >= noTotalBlocks) return;

	const RenderingBlock & b(renderingBlocks[block]);
	int xpos = b.upperLeft.x + x;
	if (xpos > b.lowerRight.x) return;
	int ypos = b.upperLeft.y + y;
	if (ypos > b.lowerRight.y) return;

	Vector2f & pixel(minmaxData[xpos + ypos*imgSize.x]);
	atomicMin(&pixel.x, b.zRange.x); atomicMax(&pixel.y, b.zRange.y);
}

__global__ void findMissingPoints_device(int *fwdProjMissingPoints, uint *noMissingPoints, const Vector2f *minmaximg,
	Vector4f *forwardProjection, float *currentDepth, Vector2i imgSize)
{
	int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

	if (x >= imgSize.x || y >= imgSize.y) return;

	int locId = x + y * imgSize.x;
	int locId2 = (int)floor((float)x / minmaximg_subsample) + (int)floor((float)y / minmaximg_subsample) * imgSize.x;

	Vector4f fwdPoint = forwardProjection[locId];
	Vector2f minmaxval = minmaximg[locId2];
	float depth = currentDepth[locId];

	bool hasPoint = false;

	__shared__ bool shouldPrefix;
	shouldPrefix = false;
	__syncthreads();

	if ((fwdPoint.w <= 0) && ((fwdPoint.x == 0 && fwdPoint.y == 0 && fwdPoint.z == 0) || (depth > 0)) && (minmaxval.x < minmaxval.y))
		//if ((fwdPoint.w <= 0) && (minmaxval.x < minmaxval.y))
	{
		shouldPrefix = true; hasPoint = true;
	}

	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device(hasPoint, noMissingPoints, blockDim.x * blockDim.y, threadIdx.x + threadIdx.y * blockDim.x);
		if (offset != -1) fwdProjMissingPoints[offset] = locId;
	}
}

__global__ void forwardProject_device(Vector4f *forwardProjection, const Vector4f *pointsRay, Vector2i imgSize, Matrix4f M,
	Vector4f projParams, float voxelSize)
{
	int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

	if (x >= imgSize.x || y >= imgSize.y) return;

	int locId = x + y * imgSize.x;
	Vector4f pixel = pointsRay[locId];

	int locId_new = forwardProjectPixel(pixel * voxelSize, M, projParams, imgSize);
	if (locId_new >= 0) forwardProjection[locId_new] = pixel;
}

} // namespace ITMLib
